#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
using namespace nvcuda;
// note: mma requires every thread within warp to participate
// otherwise it is undefined behaviour

#define WARP_SIZE 32

// pass in value LOAD/STORE 128 BITS
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

// by default cp.async instructions go to the current async group buffer
// calling cp.async.commit_group closes the curent group (push it into the queue) and starts a new one
// wait group will wait for the first n commited groups. If n=0, it will wait for all groups.
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))

// ca(cache all, L1 + L2), bytes = 4, 8, 16
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// cg(cache global, L2): bytes = 16
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// we advise L2 to prefetch the whole 128-byte line

template<const int WMMA_SIZE_M=16,
         const int WMMA_SIZE_N=16,
         const int WMMA_SIZE_K=16,
         // m16n16k16 MMA 
         const int WARPS_PER_BLOCK_M=4,
         const int WARPS_PER_BLOCK_N=2,
         // 4x2 WARP tiles
         const int WMMA_PER_WARP_M=2,
         const int WMMA_PER_WARP_N=4
         // 2x4 WMMA tiles
        >
__global__ void hgemm_m16n16k16mma4x2_wp2x4(
  half* A, half* B, half* C,
  int M, int N, int K
){
  int b_x = blockIdx.x;
  int b_y = blockIdx.y;
  constexpr int BM = WMMA_SIZE_M * WMMA_PER_WARP_M * WARPS_PER_BLOCK_M;
  constexpr int BN = WMMA_SIZE_N * WMMA_PER_WARP_N * WARPS_PER_BLOCK_N;
  constexpr int BK = WMMA_SIZE_K;
  // we take slices of 16 each time
  __shared__ half s_A[BM][BK], s_B[BK][BN];

  // determine thread and warp
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  int warp_id = tid / WARP_SIZE;
  int lane_id = tid % WARP_SIZE;
  // determine warp tile idx
  // 256 threads / 8 warps
  int warp_m = warp_id / 2; // 0, 1, 2, 3
  int warp_n = warp_id % 2; // 0, 1

  // calculate load offsets in SRAM
  // 2 thread loads a row of A
  int load_s_A_m = tid / 2;
  int load_s_A_k = (tid % 2 == 0) ? 0 : 8;
  // 16 thread loads a row of B
  int load_s_B_k = tid / 16;
  int load_s_B_n = (tid % 16) * 8;

  // calculate load offsets in GRAM
  int load_g_A_m = b_y * BM + load_s_A_m;
  int load_g_B_n = b_x * BN + load_s_B_n;

  // initialize an array of C fragments
  wmma::fragment<wmma::accumulator,
                 WMMA_SIZE_M, WMMA_SIZE_N, WMMA_SIZE_K,
                 half> frag_C[WMMA_PER_WARP_M][WMMA_PER_WARP_N];
  #pragma unroll
  for (int i=0; i<WMMA_PER_WARP_M; i++){
    #pragma unroll
    for (int j=0; j<WMMA_PER_WARP_N; j++){
      wmma::fill_fragment(frag_C[i][j], 0.0);
    }
  }

  // now head into computation
  // sliding k right + down with step size 16
  #pragma unroll
  for (int idx_k=0; idx_k<(K/WMMA_SIZE_K); idx_k++){
    // first loading in A, B fragments from GRAM
    // calculate offsets on k
    int load_g_A_k = idx_k * WMMA_SIZE_K + load_s_A_k;
    int load_g_A_off = load_g_A_m * K + load_g_A_k;
    int load_g_B_k = idx_k * WMMA_SIZE_K + load_s_B_k;
    int load_g_B_off = load_g_B_k * N + load_g_B_n;
    LDST128BITS(s_A[load_s_A_m][load_s_A_k]) = LDST128BITS(A[load_g_A_off]);
    LDST128BITS(s_B[load_s_B_k][load_s_B_n]) = LDST128BITS(B[load_g_B_off]);
    __syncthreads();

    // now initialize and load into fragments (registers) from SRAM
    wmma::fragment<wmma::matrix_a, 
                   WMMA_SIZE_M, WMMA_SIZE_N, WMMA_SIZE_K,
                   half,
                   wmma::row_major> frag_A[WMMA_PER_WARP_M];
    wmma::fragment<wmma::matrix_b, 
                   WMMA_SIZE_M, WMMA_SIZE_N, WMMA_SIZE_K,
                   half,
                   wmma::row_major> frag_B[WMMA_PER_WARP_N];
    #pragma unroll
    for (int i=0; i<WMMA_PER_WARP_M; i++){
      // for each MMA tile
      wmma::load_matrix_sync(frag_A[i], &s_A[warp_m * (WMMA_PER_WARP_M * WMMA_SIZE_M) + i * WMMA_SIZE_M][0], BK);
    }
    #pragma unroll
    for (int j=0; j<WMMA_PER_WARP_N; j++){
      wmma::load_matrix_sync(frag_B[j], &s_B[0][warp_n * (WMMA_PER_WARP_N * WMMA_SIZE_N) + j * WMMA_SIZE_N], BN);
    }

    // mma.sync
    #pragma unroll
    for (int i=0; i<WMMA_PER_WARP_M; i++){
      #pragma unroll
      for (int j=0; j<WMMA_PER_WARP_N; j++){
        wmma::mma_sync(frag_C[i][j], frag_A[i], frag_B[j], frag_C[i][j]);
      }
    }
    // before next round of loading into SRAM, sync threads blockwise
    __syncthreads();
  }
  // writing back
  #pragma unroll
  for (int i=0; i<WMMA_PER_WARP_M; i++){
    #pragma unroll
    for (int j=0; j<WMMA_PER_WARP_N; j++){
      int store_g_C_m = b_y * BM + warp_m * (WMMA_SIZE_M * WMMA_PER_WARP_M) + i * WMMA_SIZE_M;
      int store_g_C_n = b_x * BN + warp_n * (WMMA_SIZE_N * WMMA_PER_WARP_N) + j * WMMA_SIZE_N;
      wmma::store_matrix_sync(C + store_g_C_m * N + store_g_C_n, frag_C[i][j],
                              N, wmma::row_major);
    }
  }
}
        